
#include <hip/hip_runtime.h>
#include <chrono>
#include <cstdio>
#include <iostream>

#define BLOCK_SIZE 512
#define NUM_BLOCKS 45
// Maximum number of elements that can be inserted into a block queue
#define BQ_CAPACITY 2048

// Graph structure
typedef struct {
  int numNodes;  // number of nodes
  int *nodePtrs; // each int represents a position in the nodeNeighbors array:
                 // it's the first neighbor of node i (the last one is
                 // nodePtrs[i+1]-1)
  int *nodeNeighbors; // each int represents a neighbor of the node i as above
                      // explained
} Graph;

// Define a structure for the current level nodes
typedef struct {
  int *nodes;
  int numNodes;
} CurrentLevel;

// Define a structure for visited nodes
typedef struct {
  int *nodes;
  int numNodes;
} VisitedNodes;

// Define a structure for the next level nodes
typedef struct {
  int *nodes;
  int numNodes;
} NextLevel;

/**
 * @brief Loads a graph from a file.
 *
 * @param filename The name of the file containing the graph data.
 * @return Graph* A pointer to the loaded graph.
 */
Graph *loadOrderedGraphDirected(const char *filename) {
  FILE *file = fopen(filename, "r");
  if (file == NULL) {
    perror("Error opening file");
    return NULL;
  }

  // allocate memory for the graph with cudaMalloc
  Graph *graph;
  hipError_t err = hipMallocManaged(&graph, sizeof(Graph));
  if (err != hipSuccess) {
    printf("Error: %s\n", hipGetErrorString(err));
  }

  // set number of nodes reading it from the file
  fscanf(file, "%d", &graph->numNodes);

  // scan total number of edges reading it from the file
  int numEdges;
  fscanf(file, "%d", &numEdges);

  // allocate memory for nodePtrs and nodeNeighbors with cudaMalloc
  hipMallocManaged(&graph->nodePtrs, sizeof(int) * (graph->numNodes + 1));
  hipMallocManaged(&graph->nodeNeighbors, sizeof(int) * numEdges);

  int sourceNode = 0;
  int edgeIdx = 0;
  int lastSourceNode = 0;
  graph->nodePtrs[0] = 0;
  while (fscanf(file, "%d", &sourceNode) != EOF) {
    // fill nodePtrs array
    if (sourceNode != lastSourceNode) {
      graph->nodePtrs[lastSourceNode + 1] = edgeIdx;
      lastSourceNode = sourceNode;
    }

    // fill nodeNeighbors array
    fscanf(file, "%d", &graph->nodeNeighbors[edgeIdx]);
    edgeIdx++;
    // fill last element of nodePtrs array
    graph->nodePtrs[lastSourceNode + 1] = edgeIdx;
  }

  fclose(file);

  return graph;
}

Graph *loadOrderedGraphUndirected(const char *filename) {
  FILE *file = fopen(filename, "r");
  if (file == NULL) {
    perror("Error opening file");
    return NULL;
  }

  int numNodes = 0;
  fscanf(file, "%d", &numNodes);
  bool** matrix_graph = (bool**)calloc(numNodes, sizeof(bool*));

  // start by filling the matrix with zeros
  for (int i = 0; i < numNodes; i++) {
    matrix_graph[i] = (bool*)calloc(numNodes, sizeof(bool));
  }

  // scan total number of edges reading it from the file and dump it
  int numEdges;
  fscanf(file, "%d", &numEdges);

  // read the file and fill the matrix
  int sourceNode = 0;
  int destNode = 0;
  while (fscanf(file, "%d", &sourceNode) != EOF) {
    fscanf(file, "%d", &destNode);
    matrix_graph[sourceNode][destNode] = true;
    matrix_graph[destNode][sourceNode] = true;
  }

  // now we can count the number of edges
  numEdges = 0;
  for (int i = 0; i < numNodes; i++) {
    for (int j = i; j < numNodes; j++) {
      if (matrix_graph[i][j]) {
        numEdges++;
      }
    }
  }

  // allocate memory for the graph with cudaMallocManaged
  Graph *graph;
  hipMallocManaged(&graph, sizeof(Graph));

  // set number of nodes
  graph->numNodes = numNodes;

  // allocate memory for nodePtrs and nodeNeighbors with cudaMallocManaged
  hipMallocManaged(&graph->nodePtrs, sizeof(int) * (graph->numNodes + 1));
  hipMallocManaged(&graph->nodeNeighbors, sizeof(int) * numEdges);

  // fill nodePtrs and nodeNeighbors arrays
  int edgeIdx = 0;
  graph->nodePtrs[0] = 0;
  for (int i = 0; i < numNodes; i++) {
    for (int j = i; j < numNodes; j++) {
      if (matrix_graph[i][j]) {
        graph->nodeNeighbors[edgeIdx] = j;
        edgeIdx++;
      }
    }
    graph->nodePtrs[i + 1] = edgeIdx;
  }

  // free memory
  for (int i = 0; i < numNodes; i++) {
    free(matrix_graph[i]);
  }

  free(matrix_graph);

  fclose(file);

  return graph;
}

Graph *loadMTXGraph(const char *filename) {
  FILE *file = fopen(filename, "r");
  if (file == NULL) {
    return NULL;
  }

  // Read the header
  int numNodes, numEdges;
  while (1) {
    char line[1024];
    fgets(line, 1024, file);
    if (line[0] != '%') {
      sscanf(line, "%d %d %d", &numNodes, &numNodes, &numEdges);
      break;
    }
  }

  // Allocate memory for the graph structure on the GPU
  Graph *graph;
  hipMallocManaged(&graph, sizeof(Graph));
  graph->numNodes = numNodes;

  hipMallocManaged(&graph->nodePtrs, (numNodes + 1) * sizeof(int));

  hipMallocManaged(&graph->nodeNeighbors, numEdges * sizeof(int));

  // Read the data and fill the structure
  int currentRow = -1;
  int edgeCount = 0;
  graph->nodePtrs[0] = 0;
  while (!feof(file)) {
    int row, col;
    fscanf(file, "%d %d", &row, &col);

    // MTX format is 1-based, so we need to convert to 0-based
    row--;
    col--;

    while (currentRow < row) {
      currentRow++;
      graph->nodePtrs[currentRow + 1] = edgeCount;
    }
    graph->nodeNeighbors[edgeCount] = col;
    edgeCount++;
  }

  fclose(file);
  return graph;
}

// Global queuing stub
__global__ void gpu_global_queuing_kernel(int *nodePtrs, int *nodeNeighbors,
                                          int *nodeVisited, int *currLevelNodes,
                                          int *nextLevelNodes,
                                          const int numCurrLevelNodes,
                                          int *numNextLevelNodes) {

  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;

  // Iterate over the nodes in the current level. The loop stride is the total
  // number of threads.
  for (int i = idx; i < numCurrLevelNodes; i += stride) {
    // Get the node at the current index.
    int node = currLevelNodes[i];
    for (int j = nodePtrs[node]; j < nodePtrs[node + 1]; j++) {
      // Get the neighbor at the current index.
      int neighbor = nodeNeighbors[j];
      // If the neighbor has not been visited yet.
      if (nodeVisited[neighbor] == 0) {
        // Mark the neighbor as visited.
        nodeVisited[neighbor] = 1;
        // Add the neighbor to the list of nodes to visit in the next level.
        // The atomicAdd function ensures that this operation is thread-safe.
        nextLevelNodes[atomicAdd(numNextLevelNodes, 1)] = neighbor;
      }
    }
  }
}

// Block queuing stub
__global__ void gpu_block_queuing_kernel(int *nodePtrs, int *nodeNeighbors,
                                         int *nodeVisited, int *currLevelNodes,
                                         int *nextLevelNodes,
                                         const int numCurrLevelNodes,
                                         int *numNextLevelNodes) {
  //@@ Insert Block Queuing Code Here
  // Initialize shared memory queue
  // Loop over all nodes in the current level
  // Loop over all neighbors of the node
  // If the neighbor hasn't been visited yet
  // Add it to the block queue
  // If full, add it to the global queue
  // Allocate space for block queue to go into global queue
  // Store block queue in global queue
}

// Host function for global queuing invocation
void gpu_global_queuing(int *nodePtrs, int *nodeNeighbors, int *nodeVisited,
                        int *currLevelNodes, int *nextLevelNodes,
                        const int numCurrLevelNodes, int *numNextLevelNodes) {
  gpu_global_queuing_kernel<<<NUM_BLOCKS, BLOCK_SIZE>>>(
      nodePtrs, nodeNeighbors, nodeVisited, currLevelNodes, nextLevelNodes,
      numCurrLevelNodes, numNextLevelNodes);
  hipDeviceSynchronize();
}

// Host function for block queuing invocation
void gpu_block_queuing(int *nodePtrs, int *nodeNeighbors, int *nodeVisited,
                       int *currLevelNodes, int *nextLevelNodes,
                       int numCurrLevelNodes, int *numNextLevelNodes) {
  const int numBlocks = 45;
  gpu_block_queuing_kernel<<<numBlocks, BLOCK_SIZE>>>(
      nodePtrs, nodeNeighbors, nodeVisited, currLevelNodes, nextLevelNodes,
      numCurrLevelNodes, numNextLevelNodes);
}

void sequential_cpu_traversal(Graph *graph, int *nodeVisited,
                              int *currLevelNodes, int *nextLevelNodes,
                              int numCurrLevelNodes, int *numNextLevelNodes) {
  // Iterate over the nodes in the current level.
  for (int i = 0; i < numCurrLevelNodes; i++) {
    // Get the node at the current index.
    int node = currLevelNodes[i];
    // Iterate over the neighbors of the current node.
    int firstNeighbor = graph->nodePtrs[node];
    int lastNeighbor = graph->nodePtrs[node + 1];
    for (int j = firstNeighbor; j < lastNeighbor; j++) {
      // Get the neighbor at the current index.
      int neighbor = graph->nodeNeighbors[j];
      // If the neighbor has not been visited yet.
      if (nodeVisited[neighbor] == 0) {
        // Mark the neighbor as visited.
        nodeVisited[neighbor] = 1;
        // Add the neighbor to the list of nodes to visit in the next level.
        nextLevelNodes[(*numNextLevelNodes)++] = neighbor;
      }
    }
  }
}

int main() {
  const char *filename = "standard2.txt";

  Graph *graph = loadOrderedGraphUndirected(filename);

  // initialize nodeVisited, currLevelNodes, nextLevelNodes, numCurrLevelNodes,
  // numNextLevelNodes
  int *nodeVisited;
  int *currLevelNodes;
  int *nextLevelNodes;
  int numCurrLevelNodes;
  int *numNextLevelNodes;

  hipMallocManaged(&nodeVisited, sizeof(int) * graph->numNodes);
  hipMallocManaged(&currLevelNodes, sizeof(int) * graph->numNodes);
  hipMallocManaged(&nextLevelNodes, sizeof(int) * graph->numNodes);
  hipMallocManaged(&numNextLevelNodes, sizeof(int));

  hipMemset(nodeVisited, 0, sizeof(int) * graph->numNodes);
  hipMemset(currLevelNodes, 0, sizeof(int) * graph->numNodes);
  hipMemset(nextLevelNodes, 0, sizeof(int) * graph->numNodes);
  hipMemset(numNextLevelNodes, 0, sizeof(int));

  numCurrLevelNodes = 1;
  *numNextLevelNodes = 0;

  // set the source node
  currLevelNodes[0] = 0;
  nodeVisited[0] = 1;

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  auto kernel_total_time_ms = 0;

  // main loop
  while (numCurrLevelNodes > 0) {
    // start a cuda timer
    hipEventRecord(start);

    gpu_global_queuing_kernel<<<NUM_BLOCKS, BLOCK_SIZE>>>(
        graph->nodePtrs, graph->nodeNeighbors, nodeVisited, currLevelNodes,
        nextLevelNodes, numCurrLevelNodes, numNextLevelNodes);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    kernel_total_time_ms += milliseconds;

    // copy numNextLevelNodes to numCurrLevelNodes
    numCurrLevelNodes = *numNextLevelNodes;
    // reset numNextLevelNodes
    *numNextLevelNodes = 0;

    // swap currLevelNodes and nextLevelNodes
    int *tmp = currLevelNodes;
    currLevelNodes = nextLevelNodes;
    nextLevelNodes = tmp;
  }

  // print number of visited nodes
  int numVisitedNodes = 0;
  for (int i = 0; i < graph->numNodes; i++) {
    if (nodeVisited[i] == 1) {
      numVisitedNodes++;
    }
  }
  printf("Number of visited nodes: %d\n", numVisitedNodes);

  // print kernel total time
  printf("Kernel total time: %d ms\n", kernel_total_time_ms);

  return 0;
}